#include <iostream>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <map>


// __host__ __device__ unsigned char KNOWN_HASH[64] = {59, 191, 88, 123, 36, 146, 58, 115, 132, 105, 177, 202, 38, 104, 59, 214, 21, 242, 86, 124, 137, 177, 181, 43, 144, 226, 92, 197, 48, 229, 149, 153};
// __host__ __device__ unsigned char KNOWN_HASH[64] = {112, 149, 232, 60, 94, 84, 150, 4, 140, 167, 143, 41, 10, 191, 50, 14, 132, 137, 152, 231, 198, 168, 12, 172, 154, 231, 213, 213, 2, 45, 81, 47};
// __host__ __device__ unsigned char KNOWN_HASH[64] = {127, 184, 0, 44, 43, 64, 93, 197, 133, 89, 171, 248, 202, 111, 49, 215, 102, 70, 221, 158, 162, 53, 207, 64, 161, 43, 55, 171, 235, 49, 161, 65};
// __host__ __device__ unsigned char KNOWN_HASH[64] = {194, 247, 123, 195, 214, 51, 54, 218, 231, 2, 134, 132, 163, 113, 233, 60, 26, 178, 184, 2, 10, 234, 147, 113, 254, 78, 40, 27, 137, 189, 152, 236};
// __host__ __device__ unsigned char KNOWN_HASH[64] = {170, 246, 158, 40, 181, 22, 27, 160, 23, 21, 23, 222, 196, 187, 131, 44, 78, 46, 214, 157, 9, 132, 133, 91, 41, 163, 190, 20, 48, 217, 42, 28};
// __host__ __device__ unsigned char KNOWN_HASH[64] = {115, 129, 228, 97, 46, 37, 50, 178, 118, 254, 69, 105, 174, 109, 55, 24, 81, 162, 174, 65, 10, 226, 179, 159, 0, 29, 62, 106, 183, 216, 4, 25};
// __host__ __device__ unsigned char KNOWN_HASH[64] = {8, 46, 173, 223, 28, 3, 191, 159, 188, 164, 74, 16, 13, 154, 205, 220, 100, 131, 57, 147, 199, 36, 207, 46, 107, 139, 251, 29, 254, 200, 166, 187};
// __host__ __device__ unsigned char KNOWN_HASH[64] = {110, 46, 221, 45, 107, 246, 56, 150, 162, 221, 119, 179, 55, 89, 20, 199, 141, 242, 24, 195, 70, 58, 113, 115, 236, 153, 18, 149, 210, 56, 123, 223};
// __host__ __device__ unsigned char KNOWN_HASH[64] = {214, 248, 149, 169, 198, 33, 206, 60, 210, 7, 191, 8, 253, 165, 179, 57, 32, 194, 12, 143, 119, 39, 63, 37, 131, 220, 117, 218, 0, 220, 111, 32};
__device__ unsigned char KNOWN_HASH[64] = {
160, 5, 39, 204, 12, 127, 104, 209, 61, 132, 91, 14, 121, 241, 242, 183, 120, 89, 74, 124, 63, 128, 178, 205, 192, 103, 107, 40, 209, 31, 66, 93

};



// https://docs.nvidia.com/cuda/cuda-runtime-api/group__CUDART__TYPES.html
#define CHECK(val) check_cuda( (val), #val, __FILE__, __LINE__ )
void check_cuda(hipError_t res, const char *func, const char *file, const int line)
{
	if (!res)
		return ;
	std::cerr << "CUDA error = " << static_cast<unsigned int>(res);
	std::cerr << " at " << file << ":" << line << " '" << func << "' \n";
	hipDeviceReset();
	exit(1);
}


typedef unsigned char BYTE;             // 8-bit byte
typedef unsigned int  WORD;             // 32-bit word, change to "long" for 16-bit machines

typedef struct {
	BYTE data[64];
	WORD datalen;
	unsigned long long bitlen;
	WORD state[8];
} SHA256_CTX;

#define ROTLEFT(a,b) (((a) << (b)) | ((a) >> (32-(b))))
#define ROTRIGHT(a,b) (((a) >> (b)) | ((a) << (32-(b))))

#define CH(x,y,z) (((x) & (y)) ^ (~(x) & (z)))
#define MAJ(x,y,z) (((x) & (y)) ^ ((x) & (z)) ^ ((y) & (z)))
#define EP0(x) (ROTRIGHT(x,2) ^ ROTRIGHT(x,13) ^ ROTRIGHT(x,22))
#define EP1(x) (ROTRIGHT(x,6) ^ ROTRIGHT(x,11) ^ ROTRIGHT(x,25))
#define SIG0(x) (ROTRIGHT(x,7) ^ ROTRIGHT(x,18) ^ ((x) >> 3))
#define SIG1(x) (ROTRIGHT(x,17) ^ ROTRIGHT(x,19) ^ ((x) >> 10))

/**************************** VARIABLES *****************************/
__device__ static const WORD k[64] = {
	0x428a2f98,0x71374491,0xb5c0fbcf,0xe9b5dba5,0x3956c25b,0x59f111f1,0x923f82a4,0xab1c5ed5,
	0xd807aa98,0x12835b01,0x243185be,0x550c7dc3,0x72be5d74,0x80deb1fe,0x9bdc06a7,0xc19bf174,
	0xe49b69c1,0xefbe4786,0x0fc19dc6,0x240ca1cc,0x2de92c6f,0x4a7484aa,0x5cb0a9dc,0x76f988da,
	0x983e5152,0xa831c66d,0xb00327c8,0xbf597fc7,0xc6e00bf3,0xd5a79147,0x06ca6351,0x14292967,
	0x27b70a85,0x2e1b2138,0x4d2c6dfc,0x53380d13,0x650a7354,0x766a0abb,0x81c2c92e,0x92722c85,
	0xa2bfe8a1,0xa81a664b,0xc24b8b70,0xc76c51a3,0xd192e819,0xd6990624,0xf40e3585,0x106aa070,
	0x19a4c116,0x1e376c08,0x2748774c,0x34b0bcb5,0x391c0cb3,0x4ed8aa4a,0x5b9cca4f,0x682e6ff3,
	0x748f82ee,0x78a5636f,0x84c87814,0x8cc70208,0x90befffa,0xa4506ceb,0xbef9a3f7,0xc67178f2
};

/*********************** FUNCTION DEFINITIONS ***********************/
__device__ void sha256_transform(SHA256_CTX *ctx, BYTE data[])
{
	WORD a, b, c, d, e, f, g, h, i, j, t1, t2, m[64];

	for (i = 0, j = 0; i < 16; ++i, j += 4)
		m[i] = (data[j] << 24) | (data[j + 1] << 16) | (data[j + 2] << 8) | (data[j + 3]);
	for ( ; i < 64; ++i)
		m[i] = SIG1(m[i - 2]) + m[i - 7] + SIG0(m[i - 15]) + m[i - 16];

	a = ctx->state[0];
	b = ctx->state[1];
	c = ctx->state[2];
	d = ctx->state[3];
	e = ctx->state[4];
	f = ctx->state[5];
	g = ctx->state[6];
	h = ctx->state[7];

	for (i = 0; i < 64; ++i) {
		t1 = h + EP1(e) + CH(e,f,g) + k[i] + m[i];
		t2 = EP0(a) + MAJ(a,b,c);
		h = g;
		g = f;
		f = e;
		e = d + t1;
		d = c;
		c = b;
		b = a;
		a = t1 + t2;
	}

	ctx->state[0] += a;
	ctx->state[1] += b;
	ctx->state[2] += c;
	ctx->state[3] += d;
	ctx->state[4] += e;
	ctx->state[5] += f;
	ctx->state[6] += g;
	ctx->state[7] += h;
}

__device__ void sha256_init(SHA256_CTX *ctx)
{
	ctx->datalen = 0;
	ctx->bitlen = 0;
	ctx->state[0] = 0x6a09e667;
	ctx->state[1] = 0xbb67ae85;
	ctx->state[2] = 0x3c6ef372;
	ctx->state[3] = 0xa54ff53a;
	ctx->state[4] = 0x510e527f;
	ctx->state[5] = 0x9b05688c;
	ctx->state[6] = 0x1f83d9ab;
	ctx->state[7] = 0x5be0cd19;
}

__device__ void sha256_update(SHA256_CTX *ctx, const BYTE data[], size_t len)
{
	WORD i;

	for (i = 0; i < len; ++i) {
		ctx->data[ctx->datalen] = data[i];
		ctx->datalen++;
		if (ctx->datalen == 64) {
			sha256_transform(ctx, ctx->data);
			ctx->bitlen += 512;
			ctx->datalen = 0;
		}
	}
}

__device__ void sha256_final(SHA256_CTX *ctx, BYTE hash[])
{
	WORD i;

	i = ctx->datalen;

	// Pad whatever data is left in the buffer.
	if (ctx->datalen < 56) {
		ctx->data[i++] = 0x80;
		while (i < 56)
			ctx->data[i++] = 0x00;
	}
	else {
		ctx->data[i++] = 0x80;
		while (i < 64)
			ctx->data[i++] = 0x00;
		sha256_transform(ctx, ctx->data);
		memset(ctx->data, 0, 56);
	}

	// Append to the padding the total message's length in bits and transform.
	ctx->bitlen += ctx->datalen * 8;
	ctx->data[63] = ctx->bitlen;
	ctx->data[62] = ctx->bitlen >> 8;
	ctx->data[61] = ctx->bitlen >> 16;
	ctx->data[60] = ctx->bitlen >> 24;
	ctx->data[59] = ctx->bitlen >> 32;
	ctx->data[58] = ctx->bitlen >> 40;
	ctx->data[57] = ctx->bitlen >> 48;
	ctx->data[56] = ctx->bitlen >> 56;
	sha256_transform(ctx, ctx->data);

	// Since this implementation uses little endian byte ordering and SHA uses big endian,
	// reverse all the bytes when copying the final state to the output hash.
	for (i = 0; i < 4; ++i) {
		hash[i]      = (ctx->state[0] >> (24 - i * 8)) & 0x000000ff;
		hash[i + 4]  = (ctx->state[1] >> (24 - i * 8)) & 0x000000ff;
		hash[i + 8]  = (ctx->state[2] >> (24 - i * 8)) & 0x000000ff;
		hash[i + 12] = (ctx->state[3] >> (24 - i * 8)) & 0x000000ff;
		hash[i + 16] = (ctx->state[4] >> (24 - i * 8)) & 0x000000ff;
		hash[i + 20] = (ctx->state[5] >> (24 - i * 8)) & 0x000000ff;
		hash[i + 24] = (ctx->state[6] >> (24 - i * 8)) & 0x000000ff;
		hash[i + 28] = (ctx->state[7] >> (24 - i * 8)) & 0x000000ff;
	}
}

__device__ int	ft_memcmp(const void *s1, const void *s2, size_t n)
{
	const unsigned char		*str1;
	const unsigned char		*str2;

	if (s1 == s2 || n == 0)
		return (0);
	str1 = (const unsigned char *)s1;
	str2 = (const unsigned char *)s2;
	while (n--)
	{
		if (*str1 != *str2)
			return (*str1 - *str2);
		if (n)
		{
			str1++;
			str2++;
		}
	}
	return (0);
}

__global__ void	brute()
{
	unsigned char	key[16];
	unsigned char	hash[64];

	// MASK 1

	// key[0] = blockIdx.x;
	// key[1] = blockIdx.x;
	// key[2] = blockIdx.x;
	// key[3] = blockIdx.x;
	// key[4] = blockIdx.y;
	// key[5] = blockIdx.y;
	// key[6] = blockIdx.y;
	// key[7] = blockIdx.y;
	// key[8] = blockIdx.z;
	// key[9] = blockIdx.z;
	// key[10] = blockIdx.z;
	// key[11] = blockIdx.z;
	// key[12] = threadIdx.x;
	// key[13] = threadIdx.x;
	// key[14] = threadIdx.x;
	// key[15] = threadIdx.x;


key[0] = 0xf0;
key[1] = 0x8f;
key[2] = threadIdx.x;
key[3] = 0x46;
key[4] = 0x4a;
key[5] = 0xea;
key[6] = blockIdx.z;
key[7] = 0xa;
key[8] = 0xff;
key[9] = 0xf7;
key[10] = blockIdx.y;
key[11] = 0xfe;
key[12] = 0xe1;
key[13] = 0xe7;
key[14] = 0x0;
key[15] = 0xbf;


	SHA256_CTX ctx;

	sha256_init(&ctx);
	sha256_update(&ctx, key, 16);
	sha256_final(&ctx, hash);

	if (!ft_memcmp(hash, KNOWN_HASH, 32)) {
		printf("Found key: 0x%x: 0x%x, 0x%x: 0x%x, 0x%x: 0x%x, 0x%x: 0x%x,\n",
			0x08, key[6], 0x70, key[10], 0xe5, key[2], 0,0
		);
		// printf("Found key: 0x%x, 0x%x, 0x%x, 0x%x,\n",
		// 	key[0], key[2], key[6], key[14]
		// );
	}
	
}

int main(void)
{
	clock_t			start;
	clock_t			stop;

	dim3	blocks(256, 256, 256);
	dim3	threads(256);

	start = clock();

	brute<<<blocks, threads>>>();
	CHECK(hipGetLastError());
	CHECK(hipDeviceSynchronize());

	stop = clock();
	std::cerr << "Took: " << ((double)(stop - start)) / CLOCKS_PER_SEC << "\n";

	return (0);
}